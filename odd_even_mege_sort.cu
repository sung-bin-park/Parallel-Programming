#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>
#include <cmath>

using namespace std;
#define THREAD 512
#define BLOCK 4096
#define SWAP(x,y) { x = x + y; y = x - y; x = x - y; }

//odd
__global__ void odd_merge(int* data, int index) {
    int i;
    //distinct task id 
    i = threadIdx.x + blockDim.x * blockIdx.x;
   
    //odd state
    if(index % 2 == 1 && index < BLOCK * THREAD - 1){
        if (data[i] > data[i + 1])
            SWAP(data[i], data[i + 1]);
    }
}
//even
__global__ void even_merge(int* data, int index) {
    int i;
    //distinct task id 
    i = threadIdx.x + blockDim.x * blockIdx.x;

    //even state
    if (index % 2 == 0 && index < BLOCK * THREAD - 1) {
        if (data[i] > data[i + 1])
            SWAP(data[i], data[i + 1]);
    }
}

//set random data
void input_data(int* data, int size) {
    for (int i = 0; i < size; i++)
        data[i] = rand() % 100000;
}

int main() {
    //initialize
    int* dev_data;
    int size = BLOCK * THREAD;
    clock_t start;

    //allocate host memory
    int* data = (int*)malloc(size*sizeof(int));
    size_t dev_size = size * sizeof(int);

    //allocate device memory
    hipMalloc((void**)&dev_data, dev_size);

    //input data : Number of THREAD*BLOCK
    input_data(data, size);

    //finished input data and memcpy host to device
    hipMemcpy(dev_data, data, dev_size, hipMemcpyHostToDevice);

    //start odd_even_merge_sort
    start = clock();
    int half = size / 2;

    dim3 block_dim(BLOCK, 1);
    dim3 thread_dim(THREAD, 1);

    for (int i = 0; i < half; i++) {
        even_merge << <block_dim, thread_dim, dev_size >> > (dev_data, size);
        if (i != half - 1)
            odd_merge << <block_dim, thread_dim, dev_size >> > (dev_data, size);
        if (i == half - 1 && size % 2 == 0)
            odd_merge << <block_dim, thread_dim, dev_size >> > (dev_data, size);
    }
    cout << "time for "<<BLOCK*THREAD<<" datas to sort: " << (double)(clock() - start) << "ms\n";

    //ended sorting and memcpy device to host
    hipMemcpy(data, dev_data, dev_size, hipMemcpyDeviceToHost);

    //free all memory
    hipFree(dev_data);
    free(data);
    cout << "Odd_Even_Merge_Sort for CUDA";
    return 0;
}
