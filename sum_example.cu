
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<iostream>
using namespace std;

#define SIZE 1024
//call from host func (like order to gpu)
__global__ void vectoradd(int* a, int* b, int* c,int n) {
	int i = threadIdx.x;//to distinct thread
	
	for (i = 0; i < n; i++)c[i] = a[i] + b[i];
	//do sum of each thread
}

int main() {
	int* a, * b, * c;//host variable
	int* d_a, * d_b, * d_c;//device variable

	//malloc to host memory
	a = (int*)malloc(SIZE * sizeof(int));
	b = (int*)malloc(SIZE * sizeof(int));
	c = (int*)malloc(SIZE * sizeof(int));
	
	//malloc to device memory
	hipMalloc(&d_a, SIZE * sizeof(int));
	hipMalloc(&d_b, SIZE * sizeof(int));
	hipMalloc(&d_c, SIZE * sizeof(int));

	//initialize variables
	for (int i = 0; i < SIZE; i++) {
		a[i] = rand() % 1000;
		b[i] = rand() % 1000;
	}

	//memcopy to device variables
	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);
	
	//call vectoradd func using 1 block, 1024 threads
	vectoradd <<< 1, SIZE >>> (d_a, d_b, d_c, SIZE);

	//memcopy to host, save device to host 
	hipMemcpy(a, d_a, SIZE * sizeof(int), hipMemcpyDeviceToHost); 
	hipMemcpy(b, d_b, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	//print result of c variable
	for (int i = 0; i < SIZE; i++)
		cout << "c[" << i << "]=" << c[i] << "\n";
	//memory free to host
	free(a);
	free(b);
	free(c);
	//memory free to device
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;


}